
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_cuda() {
  printf("Hello World from GPU! %d\n", blockIdx.x * blockDim.x + threadIdx.x);
}

int main() {
  printf("Hello World from CPU!\n");
  hello_cuda<<<2, 2>>>();
  hipDeviceSynchronize();
}
